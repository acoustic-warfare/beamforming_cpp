#include "hip/hip_runtime.h"
#include <iostream>
#include <array>
#include <chrono>
#include <thread>
#include <vector>
#include <math.h> 
#include <numeric>
#include <functional>
#include "constants.h"
#include "filter_coefficients.h"
#include <algorithm>

using std::cout;
using std::endl;
//using namespace std;


void generate_array_r_prime(double * r_prime) {
    int elements = constants::elements;
    int column_elements = constants::column_elements;
    int row_elements = constants::row_elements;
    double uni_distance = constants::uni_distance;
    double r_a[3] = { constants::r_ax , constants::r_ay, constants::r_az };

    int element_index = 0;

      for (int i = 0; i < row_elements; i++)
      {

        for (int j = 0; j < column_elements; j++)
        { 
          r_prime[element_index] = i*uni_distance + r_a[0];
          r_prime[element_index + elements] = j*uni_distance + r_a[1];

          //std::cout << r_prime[0][element_index];

          element_index +=1;
        }
      }

      for (int i = 0; i < row_elements * column_elements; i++)
      {
        r_prime[i] -= ((double)(row_elements)*uni_distance/2) - uni_distance/2;
        r_prime[i + elements] -= ((double)(column_elements))*uni_distance/2 - uni_distance/2;
      }
}

/* void filter(double * x,int x_length,double a_0, int P) {
    double x_temp[x_length];    //Create an empty array to copy the values of x

    double temp_var;        //Sum variable
    int n;
    int i;

    for (n = 0; n < x_length; n++)
    {
        temp_var = 0;
        x_temp[n] = x[n];       //Get the past values of x and store them in x_temp
        //x[n] = std::inner_product(series1, series1 + n, series2, 0.0);
        for (i = 0; i <= n && i <= P; i++)
        {
            temp_var += filter_coefficients::filt_coeffs[0][i]*x_temp[n-i];
        }
        x[n] = 1/a_0 * temp_var;    //Store value in x, thus overwriting the past values of x, this explains the reason for x_temp
    }
} */

void generate_emulated_data(std::vector<float>& audio_data, double * r_prime) {

    // Get emulation settings
    int sample_max = (int)((constants::t_end - constants::t_start)*constants::f_sampling);
    int elements = constants::elements;
    int sources = constants::sources_N;

    // Generate the frequencies from the sources
    int max_freqs = 0;
    for (int i = 0; i < sources; i++)
    {
        if (max_freqs < constants::source_frequency_N[i])
        {
            max_freqs = constants::source_frequency_N[i];
        }
        
    }

    double frequencies[sources][max_freqs] = {0};

    for (int i = 0; i < sources; i++)
    {
        for (int j = 0; j < constants::source_frequency_N[i]; j++)
        {
            double freq_increment = (constants::source_frequency_span[i][1]-constants::source_frequency_span[i][0])/(constants::source_frequency_N[i] -1);
            frequencies[i][j] = constants::source_frequency_span[i][0] + freq_increment*j;
            //std::cout << "\n";
            //std::cout << frequencies[i][j] ;
        }        
    }
    // Generated source frequencies DONE

    // Generate emulated data
    double t = 0;

    //double r_1[3] = {0};
    double temp_signal_sample = 0;

    double theta = 0;
    double phi = 0;
    double rho = 0;
    double k = 0;

    double rho_sin_theta = 0;
    double cos_phi = 0;
    double sin_phi = 0;
    //double r_2[3] = {0};  

    double norm_factor = 0;
    double phase_offset = 0;

    double element_amplitude = 0;



    // Generate actual data
    for (int mic = 0; mic < elements ; mic++)
    {

        // Pad data with P zeros in the beginning, where P is the fitler order
        for (int j = 0; j < filter_coefficients::filter_order; j++)
        {
            audio_data.push_back(0);
        }

        double r_1[3] = {r_prime[mic],r_prime[mic + elements],r_prime[mic + 2*elements]};

        for (int i = 0; i < sample_max; i++)
        {
            t = (((double)i)/(double)constants::f_sampling);

            temp_signal_sample = 0;

            for (int source = 0; source < sources; source++)
            {
                if (constants::source_t_start[source] <= t && t < constants::source_t_end[source]) {
                    theta = constants::source_theta_deg[source]* constants::pi /180;
                    phi = constants::source_phi_deg[source]* constants::pi /180;
                    rho = constants::source_distance_away[source];
                    for (int freq_ind = 0; freq_ind < constants::source_frequency_N[source]; freq_ind++)
                    {
                        k = 2*constants::pi*frequencies[source][freq_ind]/constants::c;

                        rho_sin_theta = rho*sin(theta);
                        cos_phi = cos(phi);
                        sin_phi = sin(phi);
                        double r_2[3] = {rho_sin_theta*cos_phi,rho_sin_theta*sin_phi,rho*cos(theta) };  

                        norm_factor = sqrt( pow(r_2[0] - r_1[0],2) + pow(r_2[1] - r_1[1],2) + pow(r_2[2] - r_1[2],2) );
                        phase_offset = -k*norm_factor;

                        element_amplitude = 1/norm_factor;

                        temp_signal_sample += element_amplitude*sin(2*constants::pi*frequencies[source][freq_ind]*t + phase_offset);
                    }
                    
                }
            }
            audio_data.push_back(temp_signal_sample);
        }
        
    }
    
}
int weight_index(double frequency) {

    double lambda = constants::c/frequency;

    double lambda_rel = constants::uni_distance/lambda;
    int index;

    if (lambda_rel > 0.1581)
    {
        index = 1;

    } else if (0.156 >= lambda_rel && lambda_rel > 0.0986)
    {
        index = 3;
    } else if (0.0986 >= lambda_rel && lambda_rel > 0.085) {

        index = 5;
    } else if (0.085 >= lambda_rel && lambda_rel > 0.07) {

        index = 6;
    } else {

        index = 7;
    }

    return index;
}

void generate_weight_matrix(int * weight_matrix) {
    int elements = constants::elements;
    int config_modes = constants::available_modes;

    int columns = constants::column_elements;
    int rows = constants::row_elements;

    int element_index;

    for (int mode = 0; mode < config_modes; mode++)
    {
        int row_lim = static_cast<int>((float)(rows)/(float)(mode+1) + 0.99);

        int column_lim = static_cast<int>((float)(columns)/(float)(mode+1) + 0.99);

        int test = (int)(3/4);

        for (int i = 0; i < row_lim; i++)
        {
            for (int j = 0; j < column_lim; j++)
            {
                element_index = ((mode + 1)*(i)) * rows + (mode +1) *(j);
                weight_matrix[elements*mode + element_index] = 1;
                
            }   
        }   
    }
}

void generate_mfilter_coefficients(float * f_mega_coefficients, double * r_prime, int * weight_matrix,
    double theta, double phi) {

    int elements = constants::elements;
    int m_rows = filter_coefficients::f_bands_N * elements;
    int m_columns = filter_coefficients::filter_order +1 +2;

    double x_factor = sin(theta) * cos(phi);
    double y_factor = sin(theta) * sin(phi);

    const double a_0 = 1.0;

    const int P = filter_coefficients::filter_order;

    for (int freq_ind = 0; freq_ind < filter_coefficients::f_bands_N; freq_ind++)
    {

        // Center frequency
        double frequency = filter_coefficients::center_frequencies[freq_ind];

        // Normalized frequency
        double ny = frequency/((double)(constants::f_sampling));

        // Narrow-band wave vector 
        double k = 2*constants::pi * frequency/ constants::c;

        // Weight index 
        int w_index = weight_index(frequency)-1;

        for (int mic_ind = 0; mic_ind < elements; mic_ind++)
        {
            if (weight_matrix[elements*w_index + mic_ind] == 1)
            {                
                // FIlter coefficients for each band 
                filter_coefficients::filt_coeffs[freq_ind];

                // Row index 
                int row_index = freq_ind*elements + mic_ind;
                
                // Phase shift value theta is dependent on the frequency and the location of the element (x,y)
                double phi_0 = -k*(r_prime[mic_ind]*x_factor + r_prime[mic_ind + elements]*y_factor);

                // Calculation coefficients
                double A = sin(phi_0)/(4*constants::pi*ny*a_0);
                double B = cos(phi_0)/a_0;

                // Calculation of the mega filter coefficients!
                f_mega_coefficients[row_index*m_columns + 0] = A* filter_coefficients::filt_coeffs[freq_ind][0];
                f_mega_coefficients[row_index*m_columns + 1] = B*filter_coefficients::filt_coeffs[freq_ind][0] +  A* filter_coefficients::filt_coeffs[freq_ind][1];

                for (int i = 2; i <= P ; i++)
                {
                    f_mega_coefficients[row_index*m_columns + i] = B*filter_coefficients::filt_coeffs[freq_ind][i-1] + A*(filter_coefficients::filt_coeffs[freq_ind][i] - filter_coefficients::filt_coeffs[freq_ind][i-2]);
                }

                f_mega_coefficients[row_index*m_columns + P+1] = (B*filter_coefficients::filt_coeffs[freq_ind][P] - A*filter_coefficients::filt_coeffs[freq_ind][P-1]); 
                f_mega_coefficients[row_index*m_columns + P+2] = - A*filter_coefficients::filt_coeffs[freq_ind][P];    
  
            }   
        }   
    }
}


/*
Cuda Fir Filter for a single mic
*/
__global__ void cuFirFilter(const float *d_x, float *d_filter, float *d_y, const int filterLength, const int d_yLength){
    float sum;
    __shared__ float filt[200];
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < d_yLength*64)
    {   
        for (int l = 0; l < 64; l++)
        {
            for (int k = 0; k < 45; k++)
            {
                
                if(threadIdx.x < filterLength)
                    filt[threadIdx.x] = d_filter[threadIdx.x+filterLength*k];
                __syncthreads();

                for (int j = 0; j < d_yLength && j < filterLength; j++)
                    {
                        sum += filt[j] * d_x[i-j];
                    }

                d_y[i] = sum;
            }
        }
    }
}

/*             if(threadIdx.x < filterLength)
                filt[threadIdx.x] = d_filter[threadIdx.x+filterLength*j];
            __syncthreads();
 */

__global__ void cuFirFilterV2(const float *d_x, float *d_filter, float *d_y, const int filterLength, const int d_yLength){
    float sum;
    __shared__ float filt[202];
    int idx = 2880;
    int elems = 64;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int tidx = tid/d_yLength; 
    
    if( tid < d_yLength*elems){

        for(int j = 0; j < 64; j++){
            
            for(int k = 0; k < 45; k++){
                if(threadIdx.x < filterLength){
                    filt[threadIdx.x] = d_filter[threadIdx.x+filterLength*k+45*filterLength*j];
                }
                __syncthreads();
                
                for (int i = 0; i < d_yLength && i < filterLength; i++)
                {
                    sum += filt[i] * d_x[tid-i];
                }
                d_y[tid] = sum;
            }
        }
    }
}


int main() {
    double r_prime[3*(constants::elements)] = {0};      //initiazte r_prime full of 0s
    generate_array_r_prime(r_prime);                    //Generate r_prime     
    std::vector<float> audio_data;
    generate_emulated_data(audio_data,r_prime);
    std::cout << "Audio_matrix size: " << audio_data.size() << std::endl;

    int weight_matrix[constants::elements * constants::available_modes] = {0};
    std::cout << "weight_matrix size: " << constants::elements * constants::available_modes << std::endl;

    generate_weight_matrix(weight_matrix);

    int mega_f_size = filter_coefficients::f_bands_N * constants::elements * (filter_coefficients::filter_order + 1 + 2); 
    std::cout << "mega_f_zise: " << mega_f_size << std::endl;

    float mega_f_coefficients[mega_f_size] = {0};

    auto start3 = std::chrono::high_resolution_clock::now();
    generate_mfilter_coefficients(mega_f_coefficients,r_prime,weight_matrix,0.3,1.32);
    auto end3 = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> float_ms3 = end3 - start3;
    std::cout << "Generating Mfilter elapsed time is " << float_ms3.count() << " milliseconds" << std::endl;
    int column_size = filter_coefficients::filter_order + 3;

/*     int dataLength = 32199;
    int totDL = constants::elements * dataLength;
    int coeffs = 202;
    int dataOutLen = 32199;
    
    float *h_filteredData = new float[totDL]{0};

    float *d_data = nullptr;
    hipMalloc((void **)&d_data, totDL * sizeof(float));

    float *d_filter = nullptr;
    hipMalloc((void **)&d_filter, mega_f_size * sizeof(float));

    float *d_filteredData = nullptr;
    hipMalloc((void **)&d_filteredData, totDL * sizeof(float));

    hipMemcpy(d_data, audio_data.data(), totDL * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, mega_f_coefficients, mega_f_size * sizeof(float), hipMemcpyHostToDevice);  

    int threadsPerBlock = 256;
    int blocksPerGrid = (totDL + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "Work started: " << blocksPerGrid*threadsPerBlock << std::endl;
    
    auto start = std::chrono::high_resolution_clock::now();

    cuFirFilterV2<<<blocksPerGrid,threadsPerBlock>>>(d_data, d_filter, d_filteredData, coeffs, dataOutLen);
    
    auto end = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_filteredData, d_filteredData, totDL * sizeof(float), hipMemcpyDeviceToHost);

 */
    int dataLength = 32200;
    int coeffs = 200;
    int totDL = constants::elements * dataLength;
    int totCoeffs = coeffs*45;
    int dataOutLen = 32200;
    
    size_t NumberOfElements = sizeof(filter_coefficients::filt_coeffs[0])/sizeof(filter_coefficients::filt_coeffs[0][0]);
    std::cout << "size of : " << NumberOfElements << std::endl;

    float *h_filteredData = new float[totDL];
    float *d_data = nullptr;
    hipMalloc((void **)&d_data, totDL * sizeof(float));

    float *d_filter = nullptr;
    hipMalloc((void **)&d_filter, totCoeffs * sizeof(float));

    float *d_filteredData = nullptr;
    hipMalloc((void **)&d_filteredData, totDL * sizeof(float));

    hipMemcpy(d_data, audio_data.data(), dataLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter_coefficients::filt_coeffs, totCoeffs * sizeof(float), hipMemcpyHostToDevice);  

    int threadsPerBlock = 256;
    int blocksPerGrid = (totDL + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "Threads started: " << blocksPerGrid*threadsPerBlock << std::endl;
    
    auto start = std::chrono::high_resolution_clock::now();

    cuFirFilter<<<blocksPerGrid,threadsPerBlock>>>(d_data, d_filter, d_filteredData, coeffs, dataOutLen);
    
    auto end = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_filteredData, d_filteredData, dataOutLen * sizeof(float), hipMemcpyDeviceToHost);

 

    std::chrono::duration<double, std::milli> float_ms = end - start;
    std::cout << "CuFirFilter elapsed time is " << float_ms.count() << " milliseconds" << std::endl;
    // SINGLE DIRECTION BEAMFORMING 
    // TESTING  
    std::cout << "\n ";
    //std::cout << audio_signal_temp[16000];
    std::cout << "\n ";


    for (int i = 195; i < 206; i++)
    {
        std::cout << "Raw data: "<<  audio_data[i] << " Filtered data: ";
        std::cout << h_filteredData[i];
        std::cout << "\n ";
    }    
    for (int i = 32394; i < 32404; i++)
    {
        std::cout << "Raw data: "<<  audio_data[i] << " Filtered data: ";
        std::cout << h_filteredData[i];
        std::cout << "\n ";
    }
    for (int i = 64593; i < 64603; i++)
    {
        std::cout << "Raw data: "<<  audio_data[i] << " Filtered data: ";
        std::cout << h_filteredData[i];
        std::cout << "\n ";
    }
 
    hipFree(d_data);
    hipFree(d_filter);
    hipFree(d_filteredData);

    delete [] h_filteredData;


    return 0;
}